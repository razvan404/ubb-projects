#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "./convolution.cuh"
#include ""

__global__
void convolutionKernel(
	int* matrix, int* filter, int* output,
	int matrixDim0, int matrixDim1, int filterDim0, int filterDim1
) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= matrixDim0 || j >= matrixDim1) {
		return;
	}

	int result = 0;
	int halfFilterDim0 = filterDim0 / 2,
		halfFilterDim1 = filterDim1 / 2;
	for (int p = 0; p < filterDim0; ++p) {
		for (int q = 0; q < filterDim1; ++q) {

			int curRow = i - halfFilterDim0 + p;
			if (curRow < 0) {
				curRow = 0;
			}
			else if (curRow >= matrixDim0) {
				curRow = matrixDim0 - 1;
			}
			int curCol = j - halfFilterDim1 + q;
			if (curCol < 0) {
				curCol = 0;
			}
			else if (curCol >= matrixDim1) {
				curCol = matrixDim1 - 1;
			}

			result += matrix[curRow * matrixDim0 + curCol] * filter[p * filterDim0 + q];
		}
	}
	output[i * matrixDim0 + j] = result;
}

void convolutionInline(Matrix* matrix, Matrix* filter, int blockSqrtSize) {
	int matrixBytes = matrix->shape[0] * matrix->shape[1] * sizeof(int),
		filterBytes = filter->shape[0] * filter->shape[1] * sizeof(int);
	
	// Initialize the flatten matrices to CUDA
	int* dMatrix, * dFilter, * dOutput;
	hipMalloc((void **)&dMatrix, matrixBytes);
	hipMalloc((void **)&dFilter, filterBytes);
	hipMalloc((void **)&dOutput, matrixBytes);

	// Copy the data to the initialized matrices on CUDA
	hipMemcpy(dMatrix, matrix->data, matrixBytes, hipMemcpyHostToDevice);
	hipMemcpy(dFilter, filter->data, filterBytes, hipMemcpyHostToDevice);

	// Init the blockSize and the gridSize
	dim3 blockSize(blockSqrtSize, blockSqrtSize);
	dim3 gridSize(
		(matrix->shape[0] + blockSize.x - 1) / blockSize.x,
		(matrix->shape[1] + blockSize.y - 1) / blockSize.y
	);

	// Apply the convolution kernel
	convolutionKernel << <gridSize, blockSize >> > (
		dMatrix, dFilter, dOutput,
		matrix->shape[0], matrix->shape[1], filter->shape[0], filter->shape[1]
	);

	// Copy the data from the device to the matrix
	hipMemcpy(matrix->data, dOutput, matrixBytes, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(dMatrix);
	hipFree(dFilter);
	hipFree(dOutput);
}